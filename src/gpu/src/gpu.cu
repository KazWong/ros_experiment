
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA Kernel function to add the elements of two arrays
__global__
void GPU_add(long n, float *x, float *y)
{
/**
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
**/
/**
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
**/
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (long i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

void AssUnifiedMem(long n, float **_x, float **_y) {
  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(_x, n*sizeof(float));
  hipMallocManaged(_y, n*sizeof(float));
}

void FreeUnifiedMem(float *_x, float *_y) {
  // Free memory
  hipFree(_x);
  hipFree(_y);
}


void add(long n, float *_x, float *_y)
{
  // Run kernel on 1M elements on the CPU
//  add<<<1, 256>>>(N, x, y);
  int blockSize = 256;
  int numBlocks = (n + blockSize - 1) / blockSize;
  GPU_add<<<numBlocks, blockSize>>>(n, _x, _y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
}
