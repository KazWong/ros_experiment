
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA Kernel function to add the elements of two arrays
__global__
void add(long n, float *x, float *y)
{
/**
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
**/
/**
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
**/
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (long i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  long N = 1<<25; // 1M elements

/**
  float *x = new float[N];
  float *y = new float[N];
**/
  // Allocate Unified Memory -- accessible from CPU or GPU
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));


  // initialize x and y arrays on the host
  for (long i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the CPU
//  add<<<1, 256>>>(N, x, y);
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (long i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

/**
  // Free memory
  delete [] x;
  delete [] y;
**/
  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
